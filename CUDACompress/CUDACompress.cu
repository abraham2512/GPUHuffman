#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <limits.h>
#include "parallelHeader.h"
#define block_size 1024
#define MIN_SCRATCH_SIZE 50 * 1024 * 1024

struct huffmanTree *head_huffmanTreeNode;
struct huffmanTree huffmanTreeNode[512];
unsigned char bitSequenceConstMemory[256][255];
struct huffmanDictionary huffmanDictionary;
unsigned int constMemoryFlag = 0;

int main(int argc, char **argv){
	unsigned int i;
	unsigned int distinctCharacters, combinedHuffmanNodes, inputFileLen, frequency[256];
	unsigned char *inputFileData, bitSequenceLength = 0, bitSequence[255];
	unsigned int *compressedDataOffset, cpu_time_used;
	unsigned int integerOverflowFlag;
	FILE *inputFile, *compressedFile;
	long unsigned int mem_free, mem_total;
	long unsigned int mem_req, mem_offset, mem_data;
	int numKernelRuns;
	clock_t start, end;
	
	// check number of args
	if(argc != 3){
		printf("try with arguments InputFile and OutputFile");
		return -1;
	}
	// read input file, get inputFileLen and data
	inputFile = fopen(argv[1], "rb");
	fseek(inputFile, 0, SEEK_END);
	inputFileLen = ftell(inputFile);
	fseek(inputFile, 0, SEEK_SET);
	inputFileData = (unsigned char *)malloc(inputFileLen * sizeof(unsigned char));
	fread(inputFileData, sizeof(unsigned char), inputFileLen, inputFile);
	fclose(inputFile);
	
	// calculate run duration
	start = clock();
	
	// find the frequency of each symbols
	for (i = 0; i < 256; i++){
		frequency[i] = 0;
	}
	for (i = 0; i < inputFileLen; i++){
		frequency[inputFileData[i]]++;
	}

	// initialize nodes of huffman tree
	distinctCharacters = 0;
	for (i = 0; i < 256; i++){
		if (frequency[i] > 0){
			huffmanTreeNode[distinctCharacters].count = frequency[i];
			huffmanTreeNode[distinctCharacters].letter = i;
			huffmanTreeNode[distinctCharacters].left = NULL;
			huffmanTreeNode[distinctCharacters].right = NULL;
			distinctCharacters++;
		}
	}
	
	// build tree 
	for (i = 0; i < distinctCharacters - 1; i++){
		combinedHuffmanNodes = 2 * i;
		sortHuffmanTree(i, distinctCharacters, combinedHuffmanNodes);
		buildHuffmanTree(i, distinctCharacters, combinedHuffmanNodes);
	}
	
	if(distinctCharacters == 1){
	  head_huffmanTreeNode = &huffmanTreeNode[0];        
        }

	// build table having the bitSequence sequence and its length
	buildHuffmanDictionary(head_huffmanTreeNode, bitSequence, bitSequenceLength);
	
	// calculate memory requirements
	// GPU memory
	hipMemGetInfo(&mem_free, &mem_total);
	
	// debug
	if(1){
		printf("Free Mem: %lu\n", mem_free);		
	}

	// offset array requirements
	mem_offset = 0;
	for(i = 0; i < 256; i++){
		mem_offset += frequency[i] * huffmanDictionary.bitSequenceLength[i];
	}
	mem_offset = mem_offset % 8 == 0 ? mem_offset : mem_offset + 8 - mem_offset % 8;
	
	// other memory requirements
	mem_data = inputFileLen + (inputFileLen + 1) * sizeof(unsigned int) + sizeof(huffmanDictionary);
	
	if(mem_free - mem_data < MIN_SCRATCH_SIZE){
		printf("\nExiting : Not enough memory on GPU\nmem_free = %lu\nmin_mem_req = %lu\n", mem_free, mem_data + MIN_SCRATCH_SIZE);
		return -1;
	}
	mem_req = mem_free - mem_data - 10 * 1024 * 1024;
	numKernelRuns = ceil((double)mem_offset / mem_req);
	integerOverflowFlag = mem_req + 255 <= UINT_MAX || mem_offset + 255 <= UINT_MAX ? 0 : 1;

	// debug
	if(1){
	printf("	InputFileSize      =%u\n\
	OutputSize         =%u\n\
	NumberOfKernel     =%d\n\
	integerOverflowFlag=%d\n", inputFileLen, mem_offset/8, numKernelRuns, integerOverflowFlag);		
	}

	
	// generate data offset array
	compressedDataOffset = (unsigned int *)malloc((inputFileLen + 1) * sizeof(unsigned int));

	// launch kernel
	lauchCUDAHuffmanCompress(inputFileData, compressedDataOffset, inputFileLen, numKernelRuns, integerOverflowFlag, mem_req);

	// calculate run duration
	end = clock();
	
	// write src inputFileLen, header and compressed data to output file
	compressedFile = fopen(argv[2], "wb");
	fwrite(&inputFileLen, sizeof(unsigned int), 1, compressedFile);
	fwrite(frequency, sizeof(unsigned int), 256, compressedFile);
	fwrite(inputFileData, sizeof(unsigned char), mem_offset / 8, compressedFile);
	fclose(compressedFile);	
	
	cpu_time_used = ((end - start)) * 1000 / CLOCKS_PER_SEC;
	printf("Time taken: %d:%d s\n", cpu_time_used / 1000, cpu_time_used % 1000);
	free(inputFileData);
	free(compressedDataOffset);
	return 0;
}
